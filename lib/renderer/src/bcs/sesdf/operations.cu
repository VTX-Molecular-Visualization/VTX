#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cub/block/block_scan.cuh>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/transform_scan.h>

#include "bcs/cuda/grid.cuh"
#include "bcs/sesdf/data.cuh"
#include "bcs/sesdf/operations.cuh"

namespace bcs::sesdf
{
    void handleIntersectionSingularities( AccelerationGrid &   grid,
                                          const SesdfContext & sesContext,
                                          const float4 * const intersectionsPositions,
                                          int4 *               intersectionAtomIds,
                                          float4 *             intersectionsNeighbors )
    {
        grid.build( sesContext.intersectionWithNeighborNb, intersectionsPositions );
        auto [ numBlocks, numThreads ] = KernelConfig::From( sesContext.intersectionWithNeighborNb, 256 );
        findIntersectionsNeighbors<<<numBlocks, numThreads>>>( grid.getConfiguration(),
                                                               sesContext,
                                                               grid.getSortedPosition(),
                                                               grid.getSortedIndices(),
                                                               grid.getCellsStart(),
                                                               grid.getCellsEnd(),
                                                               intersectionsPositions,
                                                               intersectionAtomIds,
                                                               intersectionsNeighbors );
        cudaCheck( "Find Intersections Neighbors failed" );
    }

    __device__ void findCirclesInCell( const uint32_t       currentAtomIdx,
                                       const float4 &       currentAtomData,
                                       const float4 * const sortedAtomPositions,
                                       const uint32_t       startGridIndex,
                                       const uint32_t       endGridIndex,
                                       const float          probeRadius,
                                       const uint16_t       maxNeighborPerAtom,
                                       uint32_t &           currentNeighborCount,
                                       uint32_t *           neighborsIndices )
    {
        const float  ithExtendedRadius = currentAtomData.w + probeRadius;
        const float3 ithPos            = make_float3( currentAtomData );
        for ( uint32_t j = startGridIndex; j < endGridIndex; j++ )
        {
            if ( currentAtomIdx == j )
                continue;

            const float4 jthAtom           = sortedAtomPositions[ j ];
            const float3 jthPos            = make_float3( jthAtom );
            const float  jthExtendedRadius = jthAtom.w + probeRadius;

            const float dist                = length2( ithPos - jthPos );
            const float atomsExtendedRadius = ithExtendedRadius + jthExtendedRadius;
            if ( dist - atomsExtendedRadius * atomsExtendedRadius <= 1e-4f )
            {
                neighborsIndices[ currentAtomIdx * maxNeighborPerAtom + currentNeighborCount ] = j;
                currentNeighborCount++;

                if ( currentNeighborCount == maxNeighborPerAtom )
                    return;
            }
        }
    }

    __global__ void findCirclesBetweenAtoms( const GridInfo         gridInfo,
                                             const uint32_t * const cellsStart,
                                             const uint32_t * const cellsEnd,
                                             SesdfContext           sesContext )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= sesContext.atomNb )
            return;

        const float4 currentAtom = sesContext.getAtom( index );
        const int3   gridPos     = gridInfo.getGridPosition( make_float3( currentAtom ) );

        constexpr float MaxVdwRadius  = 3.48f;
        const float     gridCellRange = currentAtom.w + MaxVdwRadius + 2.f * sesContext.probeRadius;
        const int3      cellsInRange  = max( make_int3( ceilf( gridCellRange / gridInfo.cellSize ) ), make_int3( 1 ) );
        const int3      start         = max( gridPos - cellsInRange, make_int3( 0 ) );
        const int3      end           = min( gridPos + cellsInRange, gridInfo.size - 1 );

        uint32_t neighborCount = 0;
        for ( int z = start.z; z <= end.z; z++ )
        {
            for ( int y = start.y; y <= end.y; y++ )
            {
                for ( int x = start.x; x <= end.x; x++ )
                {
                    const int3     currentGridPosition = make_int3( x, y, z );
                    const uint32_t currentGridHash     = gridInfo.getHash( currentGridPosition );
                    const uint32_t startIndex          = cellsStart[ currentGridHash ];

                    if ( startIndex == AccelerationGrid::EmptyGridCellValue )
                        continue;

                    const uint32_t endIndex = cellsEnd[ currentGridHash ];

                    findCirclesInCell( index,
                                       currentAtom,
                                       sesContext.atoms,
                                       startIndex,
                                       endIndex,
                                       sesContext.probeRadius,
                                       sesContext.maxNeighborPerAtom,
                                       neighborCount,
                                       sesContext.neighborIds );
                    if ( neighborCount == sesContext.maxNeighborPerAtom )
                    {
                        printf( "Error: Too many neighbors found for atom %u.\n", blockIdx.x );

                        z = end.z + 1;
                        y = end.y + 1;
                        break;
                    }
                }
            }
        }

        sesContext.neighborNb[ index ] = neighborCount;
    }

    __global__ void computeCirclesVisibilityStatus( SesdfContext sesContext,
                                                    uint2 * __restrict__ fullCirclesData,
                                                    uint32_t * __restrict__ circlesMask )
    {
        extern __shared__ char sharedBuffer[];
        auto *                 blockAtomsData = reinterpret_cast<float4 *>( sharedBuffer );

        float4   blockAtom;
        uint16_t blockAtomCircleNb;
        uint32_t j;
        float4   jthData;
        if ( blockIdx.x < sesContext.atomNb )
        {
            blockAtom         = sesContext.getAtom( blockIdx.x );
            blockAtomCircleNb = sesContext.getNeighborNb( blockIdx.x );
            if ( threadIdx.x < blockAtomCircleNb )
            {
                j       = sesContext.getNeighborId( blockIdx.x, threadIdx.x );
                jthData = sesContext.getAtom( j );

                blockAtomsData[ threadIdx.x ] = jthData;
            }
        }

        cg::this_thread_block().sync();

        const uint32_t globalCircleIdx = blockIdx.x * sesContext.maxNeighborPerAtom + threadIdx.x;
        if ( threadIdx.x >= blockAtomCircleNb )
        {
            sesContext.visibilityStatus[ globalCircleIdx ] = 0;
            return;
        }

        uint32_t isVisible  = threadIdx.x < blockAtomCircleNb;
        uint32_t isComplete = threadIdx.x < blockAtomCircleNb;

        // Check if current circle is complete and not entirely occluded
        const CircleGeometry circle = sesContext.getCircleGeometry( blockAtom, jthData );
        for ( uint16_t kCircleIndex = 0; kCircleIndex < blockAtomCircleNb; kCircleIndex++ )
        {
            if ( threadIdx.x == kCircleIndex )
                continue;

            const float4 kData             = blockAtomsData[ kCircleIndex ];
            const float  kthExtendedRadius = kData.w + sesContext.probeRadius;

            const float3 kToCircle    = circle.center - make_float3( kData );
            const float  distToCircle = length( kToCircle );

            const float secondDistance = dot( kToCircle, circle.normal );
            const float cosTheta       = secondDistance / distToCircle;

            // Check if current circle is entirely covered by an atom
            const float sinTheta = ::sqrtf( 1.f - cosTheta * cosTheta );
            {
                const float firstDistance    = sinTheta * distToCircle + circle.radius;
                const float completeDistance = firstDistance * firstDistance + secondDistance * secondDistance;
                if ( completeDistance - kthExtendedRadius * kthExtendedRadius < 1e-4f )
                {
                    isVisible = 0;
                    break;
                }
            }

            if ( isComplete )
            {
                const float firstDistance    = ( -sinTheta ) * distToCircle + circle.radius;
                const float completeDistance = firstDistance * firstDistance + secondDistance * secondDistance;
                if ( completeDistance - kthExtendedRadius * kthExtendedRadius < 1e-4f )
                    isComplete = 0;
            }
        }

        sesContext.visibilityStatus[ globalCircleIdx ] = isVisible * ( 1 + isComplete );
        if ( isVisible == 1 && isComplete == 1 && j > blockIdx.x )
            fullCirclesData[::atomicAdd( sesContext.dFullCircleNb, 1 ) ] = make_uint2( blockIdx.x, j );

        circlesMask[ globalCircleIdx ] = static_cast<uint32_t>( !isComplete && isVisible && j > blockIdx.x );
    }

    __global__ void fillIntersections( SesdfContext     sesContext,
                                       const uint32_t * circleIntersectionStartIds,
                                       int4 *           intersectionAtomIds,
                                       uint32_t *       stencil,
                                       uint32_t *       circleIntersectionIds,
                                       float4 *         intersectionsPositions )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= *sesContext.dIntersectionNb )
            return;

        const int4 trimmedIds = intersectionAtomIds[ index ];

        const uint32_t globalIj = sesContext.trimmedToGlobal( trimmedIds.x );
        const uint32_t globalIk = sesContext.trimmedToGlobal( trimmedIds.z );

        const uint32_t i = globalIj / sesContext.maxNeighborPerAtom;
        const uint32_t j
            = sesContext.getNeighborId( i, static_cast<uint16_t>( globalIj - i * sesContext.maxNeighborPerAtom ) );
        const uint32_t k
            = sesContext.getNeighborId( i, static_cast<uint16_t>( globalIk - i * sesContext.maxNeighborPerAtom ) );

        const float4 ai    = sesContext.getAtom( i );
        const float3 aiPos = make_float3( ai );
        float        sqR2  = ai.w + sesContext.probeRadius;
        sqR2 *= sqR2;

        const float4         aj = sesContext.getAtom( j );
        const CircleGeometry ij = sesContext.getCircleGeometry( ai, aj );

        const float4         ak = sesContext.getAtom( k );
        const CircleGeometry ik = sesContext.getCircleGeometry( ai, ak );

        const float3 u = ik.normal - dot( ik.normal, ij.normal ) * ij.normal;
        const float  t = dot( ik.center - ij.center, ik.normal ) / dot( u, ik.normal );

        const float3 x1              = ij.center + t * u;
        const float  squareIToX1Dist = length2( x1 - aiPos );

        const float c = fmax( -squareIToX1Dist + sqR2, 0.f );

        const float3 n            = normalize( cross( ij.normal, ik.normal ) );
        const float3 intersection = x1 + ( sign( static_cast<float>( trimmedIds.w ) ) * ::sqrtf( c ) * n );

        intersectionsPositions[ index ] = make_float4( intersection, trimmedIds.w );

        uint32_t currentIntId = ::atomicAdd( stencil + trimmedIds.x, 1 );
        assert( currentIntId < sesContext.circlesIntersectionNb[ trimmedIds.x ] );

        uint32_t writingPosition                 = circleIntersectionStartIds[ trimmedIds.x ] + currentIntId;
        circleIntersectionIds[ writingPosition ] = index;

        currentIntId = ::atomicAdd( stencil + trimmedIds.y, 1 );
        assert( currentIntId < sesContext.circlesIntersectionNb[ trimmedIds.y ] );
        writingPosition                          = circleIntersectionStartIds[ trimmedIds.y ] + currentIntId;
        circleIntersectionIds[ writingPosition ] = index;

        currentIntId = ::atomicAdd( stencil + trimmedIds.z, 1 );
        assert( currentIntId < sesContext.circlesIntersectionNb[ trimmedIds.z ] );
        writingPosition                          = circleIntersectionStartIds[ trimmedIds.z ] + currentIntId;
        circleIntersectionIds[ writingPosition ] = index;

        intersectionAtomIds[ index ] = make_int4( i, j, k, 0 );
    }

    __global__ void findIntersectionsNeighbors( const GridInfo         gridInfo,
                                                SesdfContext           sesContext,
                                                const float4 * const   sortedIntersectionPos,
                                                const uint32_t * const sortedIntersectionIds,
                                                const uint32_t * const cellsStart,
                                                const uint32_t * const cellsEnd,
                                                const float4 * const   intersectionPositions,
                                                int4 *                 intersectionAtomIds,
                                                float4 *               intersectionNeighbors )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

        if ( index >= sesContext.intersectionWithNeighborNb )
            return;

        const uint32_t sortedIdx  = sortedIntersectionIds[ index ];
        const float3   currentPos = make_float3( sortedIntersectionPos[ index ] );
        const int3     gridPos    = gridInfo.getGridPosition( currentPos );

        const float gridCellRange = 2.f * sesContext.probeRadius;
        const int3  cellsInRange
            = max( make_int3( ceilf( make_float3( gridCellRange ) / gridInfo.cellSize ) ), make_int3( 1 ) );

        const int3 start = max( gridPos - cellsInRange, make_int3( 0 ) );
        const int3 end   = min( gridPos + cellsInRange, gridInfo.size - 1 );

        uint32_t neighborCount = 0;
        for ( int z = start.z; z <= end.z; z++ )
        {
            for ( int y = start.y; y <= end.y; y++ )
            {
                for ( int x = start.x; x <= end.x; x++ )
                {
                    const int3     currentGridPosition = make_int3( x, y, z );
                    const uint32_t currentGridHash     = gridInfo.getHash( currentGridPosition );
                    const uint32_t startIndex          = cellsStart[ currentGridHash ];

                    if ( startIndex == AccelerationGrid::EmptyGridCellValue )
                        continue;

                    const uint32_t endIndex = cellsEnd[ currentGridHash ];
                    for ( uint32_t j = startIndex; j < endIndex; j++ )
                    {
                        if ( index == j )
                            continue;

                        const float3 jthPos      = make_float3( sortedIntersectionPos[ j ] );
                        const float  currentDist = length( currentPos - jthPos );
                        if ( currentDist < gridCellRange )
                        {
                            intersectionNeighbors[ sortedIdx * sesContext.maxIntersectionNeighbors + neighborCount ]
                                = make_float4( jthPos, currentPos.x * currentPos.y * currentPos.z );

                            neighborCount++;
                            if ( neighborCount == sesContext.maxIntersectionNeighbors )
                            {
                                z = end.z + 1;
                                y = end.y + 1;
                                break;
                            }
                        }
                    }
                }
            }
        }

        intersectionAtomIds[ sortedIdx ].w = static_cast<int>( neighborCount );
    }
} // namespace bcs::sesdf
