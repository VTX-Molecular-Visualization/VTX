#include "hip/hip_runtime.h"
#include <algorithm>

#include <hip/hip_runtime.h>
#include <glm/common.hpp>
#include <hip/hip_runtime_api.h>

#include "bcs/core/molecule.hpp"
#include "bcs/cuda/grid.cuh"
#include "bcs/sesdf/data.cuh"
#include "bcs/sesdf/operations.cuh"
#include "bcs/sesdf/sesdf.hpp"

namespace bcs
{
    constexpr uint16_t Sesdf::MaxNeighborPerAtom             = BCS_SESDF_MAXNEIGHBORPERATOM;
    constexpr uint16_t Sesdf::MaxMeanIntersectionsPerCircles = 2;
    constexpr uint16_t Sesdf::MaxIntersectionsPerCircles     = 16;
    constexpr uint16_t Sesdf::MaxIntersectionNeighbors       = 32;

    Sesdf::Sesdf( ConstSpan<Vec4f> molecule, const Aabb & aabb, const float probeRadius, bool buildSurface, bool graphics ) :
        m_molecule( molecule ), m_probeRadius( probeRadius ), m_atomNb( molecule.size ), m_graphics(graphics)
    {
        constexpr float maxVdwRadius = 3.48f;

        const glm::vec3 worldOrigin = aabb.min - maxVdwRadius - m_probeRadius;
        const glm::vec3 worldSize   = glm::abs( aabb.max + maxVdwRadius + m_probeRadius - worldOrigin );

        const uint32_t  gridSize = static_cast<uint32_t>( nextPowerOfTwoValue( nextPowerOfTwoExponent( m_atomNb ) ) );
        const glm::vec3 cellSize = worldSize / static_cast<float>( gridSize );

        m_gridConfiguration.worldOrigin = make_float3( worldOrigin.x, worldOrigin.y, worldOrigin.z );
        m_gridConfiguration.cellSize    = make_float3( cellSize.x, cellSize.y, cellSize.z );
        m_gridConfiguration.size        = make_int3( static_cast<int>( gridSize ) );

        // // Pick the device with highest Gflops/s
        hipDeviceProp_t deviceProp;
        int            deviceId = gpuGetMaxGflopsDeviceId();
        cudaCheck( hipSetDevice( deviceId ) );
        cudaCheck( hipGetDeviceProperties( &deviceProp, deviceId ) );

        int isMemPoolSupported = 0;
        cudaCheck( hipDeviceGetAttribute( &isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, deviceId ) );
        assert( isMemPoolSupported );

        cudaCheck( hipSetDevice( deviceId ) );

        cudaCheck( hipHostMalloc( &m_hIntersectedCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipHostMalloc( &m_hIntersectionNb, sizeof( uint32_t ) ) );
        cudaCheck( hipHostMalloc( &m_hFullCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipHostMalloc( &m_hSegmentCount, sizeof( uint32_t ) ) );
        cudaCheck( hipHostMalloc( &m_hVisibleCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipMalloc( &m_dIntersectedCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipMalloc( &m_dIntersectionNb, sizeof( uint32_t ) ) );
        cudaCheck( hipMalloc( &m_dFullCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipMalloc( &m_dSegmentCount, sizeof( uint32_t ) ) );

        m_dAtoms         = ResultBuffer::Typed<float4>( m_atomNb, false, m_graphics);
        m_dConvexPatches = ResultBuffer::Typed<uint2>( m_atomNb, false, m_graphics );

        m_dAtomIndices            = DeviceBuffer::Typed<uint32_t>( m_atomNb );
        m_dAtomNeighborsCount     = DeviceBuffer::Typed<uint32_t>( m_atomNb );
        m_dAtomNeighborsIndices   = DeviceBuffer::Typed<uint32_t>( MaxNeighborPerAtom * m_atomNb );
        m_dCircleVisibilityStatus = DeviceBuffer::Typed<uint8_t>( MaxNeighborPerAtom * m_atomNb + 1 );
        m_dGlobalToTrimmedId      = DeviceBuffer::Typed<uint32_t>( MaxNeighborPerAtom * m_atomNb + 1 );

        hipMemPool_t memPool;
        cudaCheck( hipDeviceGetDefaultMemPool( &memPool, deviceId ) );

        constexpr uint64_t thresholdVal = std::numeric_limits<uint64_t>::max();
        cudaCheck( hipMemPoolSetAttribute( memPool, hipMemPoolAttrReleaseThreshold, (void *)&thresholdVal ) );
        m_accelerationGrid = AccelerationGrid( m_gridConfiguration );

        if ( buildSurface )
            build();
    }

    Sesdf::Sesdf( Sesdf && other )
    {
        std::swap( m_molecule, other.m_molecule );
        std::swap( m_probeRadius, other.m_probeRadius );
        std::swap( m_atomNb, other.m_atomNb );

        std::swap( m_convexPatchNb, other.m_convexPatchNb );
        std::swap( m_segmentNb, other.m_segmentNb );
        std::swap( m_intersectionNb, other.m_intersectionNb );
        std::swap( m_fullCircleNb, other.m_fullCircleNb );
        std::swap( m_sectorNb, other.m_sectorNb );

        std::swap( m_gridConfiguration, other.m_gridConfiguration );

        // Preallocated buffers
        std::swap( m_dAtomIndices, other.m_dAtomIndices );
        std::swap( m_dAtomNeighborsCount, other.m_dAtomNeighborsCount );
        std::swap( m_dAtomNeighborsIndices, other.m_dAtomNeighborsIndices );
        std::swap( m_dCircleVisibilityStatus, other.m_dCircleVisibilityStatus );
        std::swap( m_dGlobalToTrimmedId, other.m_dGlobalToTrimmedId );
        std::swap( m_accelerationGrid, other.m_accelerationGrid );

        std::swap( m_dIntersectedCircleNb, other.m_dIntersectedCircleNb );
        std::swap( m_hIntersectedCircleNb, other.m_hIntersectedCircleNb );
        std::swap( m_dIntersectionNb, other.m_dIntersectionNb );
        std::swap( m_hIntersectionNb, other.m_hIntersectionNb );
        std::swap( m_dFullCircleNb, other.m_dFullCircleNb );
        std::swap( m_hFullCircleNb, other.m_hFullCircleNb );
        std::swap( m_dSegmentCount, other.m_dSegmentCount );
        std::swap( m_hSegmentCount, other.m_hSegmentCount );
        std::swap( m_hVisibleCircleNb, other.m_hVisibleCircleNb );

        // Rendering buffers
        std::swap( m_dAtoms, other.m_dAtoms );
        std::swap( m_dIntersections, other.m_dIntersections );
        std::swap( m_dSegments, other.m_dSegments );
        std::swap( m_dConvexPatches, other.m_dConvexPatches );
        std::swap( m_dFCircleAndSectors, other.m_dFCircleAndSectors );
    }

    Sesdf & Sesdf::operator=( Sesdf && other )
    {
        std::swap( m_molecule, other.m_molecule );
        std::swap( m_probeRadius, other.m_probeRadius );
        std::swap( m_atomNb, other.m_atomNb );

        std::swap( m_convexPatchNb, other.m_convexPatchNb );
        std::swap( m_segmentNb, other.m_segmentNb );
        std::swap( m_intersectionNb, other.m_intersectionNb );
        std::swap( m_fullCircleNb, other.m_fullCircleNb );
        std::swap( m_sectorNb, other.m_sectorNb );

        std::swap( m_gridConfiguration, other.m_gridConfiguration );

        // Preallocated buffers
        std::swap( m_dAtomIndices, other.m_dAtomIndices );
        std::swap( m_dAtomNeighborsCount, other.m_dAtomNeighborsCount );
        std::swap( m_dAtomNeighborsIndices, other.m_dAtomNeighborsIndices );
        std::swap( m_dCircleVisibilityStatus, other.m_dCircleVisibilityStatus );
        std::swap( m_dGlobalToTrimmedId, other.m_dGlobalToTrimmedId );
        std::swap( m_accelerationGrid, other.m_accelerationGrid );

        std::swap( m_dIntersectedCircleNb, other.m_dIntersectedCircleNb );
        std::swap( m_hIntersectedCircleNb, other.m_hIntersectedCircleNb );
        std::swap( m_dIntersectionNb, other.m_dIntersectionNb );
        std::swap( m_hIntersectionNb, other.m_hIntersectionNb );
        std::swap( m_dFullCircleNb, other.m_dFullCircleNb );
        std::swap( m_hFullCircleNb, other.m_hFullCircleNb );
        std::swap( m_dSegmentCount, other.m_dSegmentCount );
        std::swap( m_hSegmentCount, other.m_hSegmentCount );
        std::swap( m_hVisibleCircleNb, other.m_hVisibleCircleNb );

        // Rendering buffers
        std::swap( m_dAtoms, other.m_dAtoms );
        std::swap( m_dIntersections, other.m_dIntersections );
        std::swap( m_dSegments, other.m_dSegments );
        std::swap( m_dConvexPatches, other.m_dConvexPatches );
        std::swap( m_dFCircleAndSectors, other.m_dFCircleAndSectors );

        return *this;
    }

    Sesdf::~Sesdf()
    {
        cudaCheck( hipHostFree( m_hIntersectedCircleNb ) );
        cudaCheck( hipHostFree( m_hIntersectionNb ) );
        cudaCheck( hipHostFree( m_hFullCircleNb ) );
        cudaCheck( hipHostFree( m_hSegmentCount ) );
        cudaCheck( hipHostFree( m_hVisibleCircleNb ) );
        cudaCheck( hipFree( m_dIntersectedCircleNb ) );
        cudaCheck( hipFree( m_dIntersectionNb ) );
        cudaCheck( hipFree( m_dFullCircleNb ) );
        cudaCheck( hipFree( m_dSegmentCount ) );
    }

    void Sesdf::build()
    {
        {
            sesdf::SesdfContext sesContext {};
            sesContext.probeRadius                = m_probeRadius;
            sesContext.atomNb                     = m_atomNb;
            sesContext.atoms                      = m_dAtoms.get<float4>();
            sesContext.sortedToInitialIndices     = m_dAtomIndices.get<uint32_t>();
            sesContext.neighborNb                 = m_dAtomNeighborsCount.get<uint32_t>();
            sesContext.neighborIds                = m_dAtomNeighborsIndices.get<uint32_t>();
            sesContext.visibilityStatus           = m_dCircleVisibilityStatus.get<uint8_t>();
            sesContext.maxNeighborPerAtom         = MaxNeighborPerAtom;
            sesContext.maxIntersectionsPerCircles = MaxMeanIntersectionsPerCircles;
            sesContext.maxIntersectionNeighbors   = MaxIntersectionNeighbors;
            sesContext.globalToTrimmedId          = m_dGlobalToTrimmedId.get<uint32_t>();

            sesContext.dIntersectedCircleNb = m_dIntersectedCircleNb;
            hipMemset( sesContext.dIntersectedCircleNb, 0, sizeof( uint32_t ) );
            sesContext.hIntersectedCircleNb  = m_hIntersectedCircleNb;
            *sesContext.hIntersectedCircleNb = 0;

            sesContext.dIntersectionNb = m_dIntersectionNb;
            hipMemset( sesContext.dIntersectionNb, 0, sizeof( uint32_t ) );
            sesContext.hIntersectionNb  = m_hIntersectionNb;
            *sesContext.hIntersectionNb = 0;

            sesContext.dFullCircleNb = m_dFullCircleNb;
            hipMemset( sesContext.dFullCircleNb, 0, sizeof( uint32_t ) );
            sesContext.hFullCircleNb  = m_hFullCircleNb;
            *sesContext.hFullCircleNb = 0;

            sesContext.dSegmentCount = m_dSegmentCount;
            hipMemset( sesContext.dSegmentCount, 0, sizeof( uint32_t ) );
            sesContext.hSegmentCount  = m_hSegmentCount;
            *sesContext.hSegmentCount = 0;

            sesContext.hVisibleCircleNb  = m_hVisibleCircleNb;
            *sesContext.hVisibleCircleNb = 0;

            // #1: CPU => GPU
            mmemcpy<MemcpyType::HostToDevice>(
                m_dAtoms.get<float4>(), reinterpret_cast<const float4 *>( m_molecule.ptr ), m_atomNb );

            // #2: Find Circles
            auto [ fCircleAndSectors, trimmedToGlobalId ] = sesdf::findCircles<MaxNeighborPerAtom>(
                m_accelerationGrid, sesContext, m_dConvexPatches.get<uint2>(), m_graphics );

            sesContext.trimmedToGlobalId = trimmedToGlobalId.get<uint32_t>();
            m_dFCircleAndSectors         = std::move( fCircleAndSectors );
            m_convexPatchNb              = m_atomNb;
            m_fullCircleNb               = *sesContext.hFullCircleNb;
            m_sectorNb = ( m_dFCircleAndSectors.size() - m_fullCircleNb * sizeof( uint2 ) ) / sizeof( uint32_t );

            if ( *sesContext.hIntersectedCircleNb > 0 )
            {
                // #3: Find intersections
                auto circlesIntersectionsNb = DeviceBuffer::Typed<uint32_t>( *sesContext.hIntersectedCircleNb, true );
                sesContext.circlesIntersectionNb = circlesIntersectionsNb.get<uint32_t>();

                auto [ dIntersections, startandIdList ] = sesdf::findIntersections<MaxNeighborPerAtom>( sesContext, m_graphics );

                m_intersectionNb             = *sesContext.hIntersectionNb;
                m_intersectionWithNeighborNb = sesContext.intersectionWithNeighborNb;
                m_dIntersections             = std::move( dIntersections );

                sesContext.circlesIntersectionStartId = startandIdList.get<uint32_t>();
                sesContext.circlesIntersectionIds = startandIdList.get<uint32_t>() + *sesContext.hIntersectedCircleNb;

                // #4: Build P_t
                auto segmentsData = sesdf::buildConicPatches<MaxIntersectionsPerCircles>(
                    sesContext,
                    m_dIntersections.get<const float4>( m_intersectionNb * sizeof( int4 ) ),
                    m_dIntersections.get<const int4>(),
                    m_graphics );

                m_dSegments = std::move( segmentsData );
                m_segmentNb = *sesContext.hSegmentCount;

                // #5: Build P_- neighbors
                sesdf::handleIntersectionSingularities(
                    m_accelerationGrid,
                    sesContext,
                    m_dIntersections.get<const float4>( m_intersectionNb * sizeof( int4 ) ),
                    m_dIntersections.get<int4>(),
                    m_dIntersections.get<float4>( m_intersectionNb * ( sizeof( int4 ) + sizeof( float4 ) ) ) );

                m_dIntersections.unmap();
                m_dSegments.unmap();
                m_dConvexPatches.unmap();
                m_dFCircleAndSectors.unmap();
            }
        }
    }

    sesdf::SesdfGraphics Sesdf::getGraphics() const
    {
        assert(m_graphics && "Computation has not been configured for graphics!");

        return { m_atomNb,
                 { m_dAtoms.getId(), m_atomNb * sizeof( float4 ) },

                 // Convex patch data
                 m_convexPatchNb,
                 { m_dConvexPatches.getId(), m_atomNb * sizeof( uint2 ) },

                 m_sectorNb,
                 { m_dFCircleAndSectors.getId(),
                   m_fullCircleNb * sizeof( uint2 ) + ( ( m_fullCircleNb * sizeof( uint2 ) ) % sizeof( float4 ) ),
                   m_sectorNb * sizeof( float4 ) },

                 // Toroidal patches
                 m_segmentNb,
                 { m_dSegments.getId(), m_segmentNb * sizeof( uint4 ) },

                 m_fullCircleNb,
                 { m_dFCircleAndSectors.getId(), m_fullCircleNb * sizeof( uint2 ) },

                 // Concave patches
                 m_intersectionNb,
                 { m_dIntersections.getId(), m_intersectionNb * sizeof( float4 ), m_intersectionNb * sizeof( int4 ) },
                 { m_dIntersections.getId(), m_intersectionNb * sizeof( int4 ) },
                 { m_dIntersections.getId(),
                   m_intersectionNb * ( sizeof( int4 ) + sizeof( float4 ) ),
                   m_intersectionWithNeighborNb * MaxIntersectionNeighbors * sizeof( float4 ) },

                 m_probeRadius,
                 MaxIntersectionNeighbors };
    }


    sesdf::SesdfData Sesdf::getData()
    {
        float4 * dAtoms         = m_dAtoms.get<float4>();
        uint2 *  dConvexPatches = m_dConvexPatches.get<uint2>();

        uint32_t sectorPadding
            = m_fullCircleNb * sizeof( uint2 ) + ( ( m_fullCircleNb * sizeof( uint2 ) ) % sizeof( float4 ) );
        float4 * dSectors = reinterpret_cast<float4 *>( m_dFCircleAndSectors.get() + sectorPadding );

        uint4 * dSegments    = m_dSegments.get<uint4>();
        uint2 * dFullCircles = m_dFCircleAndSectors.get<uint2>();

        float4 * concavePatchPositions
            = reinterpret_cast<float4 *>( m_dIntersections.get() + m_intersectionNb * sizeof( float4 ) );
        float4 * concavePatcheNeighbors = reinterpret_cast<float4 *>(
            m_dIntersections.get() + m_intersectionNb * ( sizeof( int4 ) + sizeof( float4 ) ) );

        return { m_atomNb,
                 dAtoms,

                 // Convex patch data
                 m_convexPatchNb,
                 dConvexPatches,

                 m_sectorNb,
                 dSectors,

                 // Toroidal patches
                 m_segmentNb,
                 dSegments,

                 m_fullCircleNb,
                 dFullCircles,

                 // Concave patches
                 m_intersectionNb,
                 concavePatchPositions,
                 m_dIntersections.get<int4>(),
                 concavePatcheNeighbors,

                 m_probeRadius,
                 MaxIntersectionNeighbors };
    }

    uint16_t Sesdf::getMaxNeighborPerAtom() const { return MaxNeighborPerAtom; }
    uint16_t Sesdf::getMaxIntersectionsPerCircles() const { return MaxIntersectionsPerCircles; }
    uint16_t Sesdf::getMaxIntersectionNeighbors() const { return MaxIntersectionNeighbors; }

    std::vector<uint32_t> Sesdf::getAtomIndices() { return m_dAtomIndices.toHost<uint32_t>(); }

    DeviceBuffer & Sesdf::getDAtomIndices() { return m_dAtomIndices; }
    DeviceBuffer & Sesdf::getDAtomNeighborsCount() { return m_dAtomNeighborsCount; }
    DeviceBuffer & Sesdf::getDAtomNeighborsIndices() { return m_dAtomNeighborsIndices; }
    DeviceBuffer & Sesdf::getDCircleVisibilityStatus() { return m_dCircleVisibilityStatus; }
    DeviceBuffer & Sesdf::getDGlobalToTrimmedId() { return m_dGlobalToTrimmedId; }

} // namespace bcs
