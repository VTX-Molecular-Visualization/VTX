#include "hip/hip_runtime.h"
#include <algorithm>

#include <hip/hip_runtime.h>
#include <glm/common.hpp>
#include <hip/hip_runtime_api.h>

#include "bcs/core/molecule.hpp"
#include "bcs/cuda/grid.cuh"
#include "bcs/ssesdf/data.cuh"
#include "bcs/ssesdf/operations.cuh"
#include "bcs/ssesdf/ssesdf.hpp"

namespace bcs
{
    constexpr uint16_t Ssesdf::MaxNeighborPerAtom         = BCS_SSESDF_MAXNEIGHBORPERATOM;
    constexpr uint16_t Ssesdf::MaxIntersectionsPerCircles = 2;
    constexpr uint16_t Ssesdf::MaxIntersectionNeighbors   = 32;

    Ssesdf::Ssesdf( ConstSpan<Vec4f> molecule, const Aabb & aabb, const float probeRadius, bool buildSurface, bool graphics ) :
        m_molecule( molecule ), m_probeRadius( probeRadius ), m_atomNb( molecule.size ), m_graphics(graphics)
    {
        constexpr float maxVdwRadius = 3.48f;

        const glm::vec3 worldOrigin = aabb.min - maxVdwRadius - m_probeRadius;
        const glm::vec3 worldSize   = glm::abs( aabb.max + maxVdwRadius + m_probeRadius - worldOrigin );

        const uint32_t  gridSize = static_cast<uint32_t>( nextPowerOfTwoValue( nextPowerOfTwoExponent( m_atomNb ) ) );
        const glm::vec3 cellSize = worldSize / static_cast<float>( gridSize );

        m_gridConfiguration.worldOrigin = make_float3( worldOrigin.x, worldOrigin.y, worldOrigin.z );
        m_gridConfiguration.cellSize    = make_float3( cellSize.x, cellSize.y, cellSize.z );
        m_gridConfiguration.size        = make_int3( static_cast<int>( gridSize ) );

        // // Pick the device with highest Gflops/s
        hipDeviceProp_t deviceProp;
        int            deviceId = gpuGetMaxGflopsDeviceId();
        cudaCheck( hipSetDevice( deviceId ) );
        cudaCheck( hipGetDeviceProperties( &deviceProp, deviceId ) );

        int isMemPoolSupported = 0;
        cudaCheck( hipDeviceGetAttribute( &isMemPoolSupported, hipDeviceAttributeMemoryPoolsSupported, deviceId ) );
        assert( isMemPoolSupported );

        cudaCheck( hipSetDevice( deviceId ) );

        cudaCheck( hipHostMalloc( &m_hIntersectionNb, sizeof( uint32_t ) ) );
        cudaCheck( hipHostMalloc( &m_hVisibleCircleNb, sizeof( uint32_t ) ) );
        cudaCheck( hipMalloc( &m_dIntersectionNb, sizeof( uint32_t ) ) );

        m_dAtoms = ResultBuffer::Typed<float4>( m_atomNb, false, m_graphics );

        m_dAtomIndices            = DeviceBuffer::Typed<uint32_t>( m_atomNb );
        m_dAtomNeighborsCount     = DeviceBuffer::Typed<uint32_t>( m_atomNb );
        m_dAtomNeighborsIndices   = DeviceBuffer::Typed<uint32_t>( MaxNeighborPerAtom * m_atomNb );
        m_dCircleVisibilityStatus = DeviceBuffer::Typed<uint8_t>( MaxNeighborPerAtom * m_atomNb + 1 );

        hipMemPool_t memPool;
        cudaCheck( hipDeviceGetDefaultMemPool( &memPool, deviceId ) );

        constexpr uint64_t thresholdVal = std::numeric_limits<uint64_t>::max();
        cudaCheck( hipMemPoolSetAttribute( memPool, hipMemPoolAttrReleaseThreshold, (void *)&thresholdVal ) );
        m_accelerationGrid = AccelerationGrid( m_gridConfiguration );

        if ( buildSurface )
            build();
    }

    Ssesdf::Ssesdf( Ssesdf && other )
    {
        std::swap( m_molecule, other.m_molecule );
        std::swap( m_probeRadius, other.m_probeRadius );
        std::swap( m_atomNb, other.m_atomNb );

        std::swap( m_intersectionNb, other.m_intersectionNb );
        std::swap( m_intersectionWithNeighborNb, other.m_intersectionWithNeighborNb );
        std::swap( m_circleNb, other.m_circleNb );

        std::swap( m_gridConfiguration, other.m_gridConfiguration );

        // Preallocated buffers
        std::swap( m_dAtomIndices, other.m_dAtomIndices );
        std::swap( m_dAtomNeighborsCount, other.m_dAtomNeighborsCount );
        std::swap( m_dAtomNeighborsIndices, other.m_dAtomNeighborsIndices );
        std::swap( m_dCircleVisibilityStatus, other.m_dCircleVisibilityStatus );
        std::swap( m_accelerationGrid, other.m_accelerationGrid );

        std::swap( m_dIntersectionNb, other.m_dIntersectionNb );
        std::swap( m_hIntersectionNb, other.m_hIntersectionNb );
        std::swap( m_hVisibleCircleNb, other.m_hVisibleCircleNb );

        // Rendering buffers
        std::swap( m_dAtoms, other.m_dAtoms );
        std::swap( m_dIntersections, other.m_dIntersections );
    }

    Ssesdf & Ssesdf::operator=( Ssesdf && other )
    {
        std::swap( m_molecule, other.m_molecule );
        std::swap( m_probeRadius, other.m_probeRadius );
        std::swap( m_atomNb, other.m_atomNb );

        std::swap( m_intersectionNb, other.m_intersectionNb );
        std::swap( m_intersectionWithNeighborNb, other.m_intersectionWithNeighborNb );
        std::swap( m_circleNb, other.m_circleNb );

        std::swap( m_gridConfiguration, other.m_gridConfiguration );

        // Preallocated buffers
        std::swap( m_dAtomIndices, other.m_dAtomIndices );
        std::swap( m_dAtomNeighborsCount, other.m_dAtomNeighborsCount );
        std::swap( m_dAtomNeighborsIndices, other.m_dAtomNeighborsIndices );
        std::swap( m_dCircleVisibilityStatus, other.m_dCircleVisibilityStatus );
        std::swap( m_accelerationGrid, other.m_accelerationGrid );

        std::swap( m_dIntersectionNb, other.m_dIntersectionNb );
        std::swap( m_hIntersectionNb, other.m_hIntersectionNb );
        std::swap( m_hVisibleCircleNb, other.m_hVisibleCircleNb );

        // Rendering buffers
        std::swap( m_dAtoms, other.m_dAtoms );
        std::swap( m_dIntersections, other.m_dIntersections );

        return *this;
    }

    Ssesdf::~Ssesdf()
    {
        cudaCheck( hipHostFree( m_hIntersectionNb ) );
        cudaCheck( hipHostFree( m_hVisibleCircleNb ) );
        cudaCheck( hipFree( m_dIntersectionNb ) );
    }

    void Ssesdf::build()
    {
        {
            ssesdf::SsesdfContext sesContext {};
            sesContext.probeRadius                = m_probeRadius;
            sesContext.atomNb                     = m_atomNb;
            sesContext.atoms                      = m_dAtoms.get<float4>();
            sesContext.sortedToInitialIndices     = m_dAtomIndices.get<uint32_t>();
            sesContext.neighborNb                 = m_dAtomNeighborsCount.get<uint32_t>();
            sesContext.neighborIds                = m_dAtomNeighborsIndices.get<uint32_t>();
            sesContext.visibilityStatus           = m_dCircleVisibilityStatus.get<uint8_t>();
            sesContext.maxNeighborPerAtom         = MaxNeighborPerAtom;
            sesContext.maxIntersectionsPerCircles = MaxIntersectionsPerCircles;
            sesContext.maxIntersectionNeighbors   = MaxIntersectionNeighbors;

            sesContext.dIntersectionNb = m_dIntersectionNb;
            hipMemset( sesContext.dIntersectionNb, 0, sizeof( uint32_t ) );
            sesContext.hIntersectionNb  = m_hIntersectionNb;
            *sesContext.hIntersectionNb = 0;

            sesContext.hVisibleCircleNb  = m_hVisibleCircleNb;
            *sesContext.hVisibleCircleNb = 0;

            // #1: CPU => GPU
            mmemcpy<MemcpyType::HostToDevice>(
                m_dAtoms.get<float4>(), reinterpret_cast<const float4 *>( m_molecule.ptr ), m_atomNb );

            // #2: Find Circles
            ssesdf::findCircles( m_accelerationGrid, sesContext );
            m_circleNb = *sesContext.hVisibleCircleNb;

            if ( m_circleNb > 0 )
            {
                // #3: Find intersections
                auto [ dCircles, dIntersections ] = ssesdf::findIntersections<MaxNeighborPerAtom>( sesContext, m_graphics );
                m_dCircles                        = std::move( dCircles );
                m_circleNb                        = m_dCircles.size() / ( sizeof( float4 ) * 3 );

                m_intersectionNb             = *sesContext.hIntersectionNb;
                m_intersectionWithNeighborNb = sesContext.intersectionWithNeighborNb;
                m_dIntersections             = std::move( dIntersections );

                // #4: Build P_- neighbors
                ssesdf::handleIntersectionSingularities( m_accelerationGrid,
                                                         sesContext,
                                                         m_dIntersections.get<float4>(),
                                                         m_dIntersections.get<int4>() + m_intersectionNb,
                                                         m_dIntersections.get<float4>() + m_intersectionNb * 2 );

                m_dAtoms.unmap();
                m_dCircles.unmap();
                m_dIntersections.unmap();
            }
        }
    }

    ssesdf::SsesdfGraphics Ssesdf::getGraphics() const
    {
        return { m_atomNb,
                 { m_dAtoms.getId(), 0, m_atomNb * sizeof( float4 ) },

                 // Toroidal patches
                 m_circleNb,
                 { m_dCircles.getId(), 0, m_circleNb * sizeof( float4 ) },                             // circlePosition
                 { m_dCircles.getId(), m_circleNb * sizeof( float4 ), m_circleNb * sizeof( float4 ) }, // circleAxis
                 { m_dCircles.getId(), m_circleNb * sizeof( float4 ) * 2, m_circleNb * sizeof( float4 ) }, // circleVs

                 // Concave patches
                 m_intersectionNb,
                 { m_dIntersections.getId(), 0, m_intersectionNb * sizeof( float4 ) },
                 { m_dIntersections.getId(), m_intersectionNb * sizeof( int4 ), m_intersectionNb * sizeof( int4 ) },
                 { m_dIntersections.getId(),
                   m_intersectionNb * ( sizeof( int4 ) + sizeof( float4 ) ),
                   m_intersectionWithNeighborNb * MaxIntersectionNeighbors * sizeof( float4 ) },

                 m_probeRadius,
                 MaxIntersectionNeighbors };
    }

    ssesdf::SsesdfData Ssesdf::getData()
    {
        float4 * dAtoms = m_dAtoms.get<float4>();
        return { m_atomNb,
                 dAtoms,

                 // Toroidal patches
                 m_circleNb,
                 m_dCircles.get<float4>(),
                 m_dCircles.get<float4>( m_circleNb * sizeof( float4 ) ),
                 m_dCircles.get<float4>( m_circleNb * sizeof( float4 ) * 2 ),

                 // Concave patches
                 m_intersectionNb,
                 m_dIntersections.get<float4>(),
                 m_dIntersections.get<int4>( m_intersectionNb * sizeof( float4 ) ),
                 m_dIntersections.get<float4>( m_intersectionNb * sizeof( float4 ) * 2 ),

                 m_probeRadius,
                 MaxIntersectionNeighbors };
    }

    uint16_t Ssesdf::getMaxNeighborPerAtom() const { return MaxNeighborPerAtom; }
    uint16_t Ssesdf::getMaxIntersectionsPerCircles() const { return MaxIntersectionsPerCircles; }
    uint16_t Ssesdf::getMaxIntersectionNeighbors() const { return MaxIntersectionNeighbors; }

    std::vector<uint32_t> Ssesdf::getAtomIndices() { return m_dAtomIndices.toHost<uint32_t>(); }

    DeviceBuffer & Ssesdf::getDAtomIndices() { return m_dAtomIndices; }
    DeviceBuffer & Ssesdf::getDAtomNeighborsCount() { return m_dAtomNeighborsCount; }
    DeviceBuffer & Ssesdf::getDAtomNeighborsIndices() { return m_dAtomNeighborsIndices; }
    DeviceBuffer & Ssesdf::getDCircleVisibilityStatus() { return m_dCircleVisibilityStatus; }

} // namespace bcs
