#include "hip/hip_runtime.h"
#include <cub/block/block_scan.cuh>
#include <hip/hip_runtime.h>
#include <>
#include <thrust/sort.h>
#include <thrust/transform_scan.h>

#include "bcs/cuda/grid.cuh"
#include "bcs/ssesdf/data.cuh"
#include "bcs/ssesdf/operations.cuh"

namespace bcs::ssesdf
{
    template<class Type>
    struct IsNonZero : thrust::unary_function<Type, Type>
    {
        __host__ __device__ Type operator()( Type x ) { return static_cast<Type>( x != 0 ); }
    };

    template<class Type>
    struct IsIntersected : thrust::unary_function<Type, Type>
    {
        __host__ __device__ Type operator()( Type x ) { return static_cast<Type>( x == 1 ); }
    };

    void findCircles( AccelerationGrid & grid, SsesdfContext & sesContext )
    {
        {
            grid.build( sesContext.atomNb, sesContext.atoms );

            copy( sesContext.atoms, grid.getSortedPosition(), sesContext.atomNb );
            copy( sesContext.sortedToInitialIndices, grid.getSortedIndices(), sesContext.atomNb );

            auto [ numBlocks, numThreads ] = KernelConfig::From( sesContext.atomNb, sesContext.maxNeighborPerAtom );
            findCirclesBetweenAtoms<<<numBlocks, numThreads>>>(
                grid.getConfiguration(), grid.getCellsStart(), grid.getCellsEnd(), sesContext );
            cudaCheck( "Circles evaluation failed" );
        }

        computeCirclesVisibilityStatus<<<sesContext.atomNb,
                                         sesContext.maxNeighborPerAtom,
                                         sesContext.maxNeighborPerAtom * sizeof( float4 )>>>( sesContext );
        cudaCheck( "Circles visibility evaluation failed" );

        *sesContext.hVisibleCircleNb
            = thrust::count_if( thrust::device,
                                sesContext.visibilityStatus,
                                sesContext.visibilityStatus + sesContext.atomNb * sesContext.maxNeighborPerAtom,
                                IsNonZero<uint32_t>() );

        sesContext.intersectedCircleNb
            = thrust::count_if( thrust::device,
                                sesContext.visibilityStatus,
                                sesContext.visibilityStatus + sesContext.atomNb * sesContext.maxNeighborPerAtom,
                                IsNonZero<uint32_t>() );
    }

    void handleIntersectionSingularities( AccelerationGrid &    grid,
                                          const SsesdfContext & sesContext,
                                          float4 *              intersectionsPositions,
                                          int4 *                intersectionAtomIds,
                                          float4 *              intersectionsNeighbors )
    {
        grid.build( sesContext.intersectionWithNeighborNb, intersectionsPositions );
        auto [ numBlocks, numThreads ] = KernelConfig::From( sesContext.intersectionWithNeighborNb, 256 );
        findIntersectionsNeighbors<<<numBlocks, numThreads>>>( grid.getConfiguration(),
                                                               sesContext,
                                                               grid.getSortedPosition(),
                                                               grid.getSortedIndices(),
                                                               grid.getCellsStart(),
                                                               grid.getCellsEnd(),
                                                               intersectionsPositions,
                                                               intersectionAtomIds,
                                                               intersectionsNeighbors );
        cudaCheck( "Find Intersections Neighbors failed" );
    }

    __device__ void findCirclesInCell( const uint32_t       currentAtomIdx,
                                       const float4 &       currentAtomData,
                                       const float4 * const sortedAtomPositions,
                                       const uint32_t       startGridIndex,
                                       const uint32_t       endGridIndex,
                                       const float          probeRadius,
                                       const uint16_t       maxNeighborPerAtom,
                                       uint32_t &           currentNeighborCount,
                                       uint32_t *           neighborsIndices )
    {
        const float  ithExtendedRadius = currentAtomData.w + probeRadius;
        const float3 ithPos            = make_float3( currentAtomData );
        for ( uint32_t j = startGridIndex; j < endGridIndex; j++ )
        {
            if ( currentAtomIdx == j )
                continue;

            const float4 jthAtom           = sortedAtomPositions[ j ];
            const float3 jthPos            = make_float3( jthAtom );
            const float  jthExtendedRadius = jthAtom.w + probeRadius;

            const float dist                = length2( ithPos - jthPos );
            const float atomsExtendedRadius = ithExtendedRadius + jthExtendedRadius;
            if ( dist - atomsExtendedRadius * atomsExtendedRadius <= 1e-4f )
            {
                neighborsIndices[ currentAtomIdx * maxNeighborPerAtom + currentNeighborCount ] = j;
                currentNeighborCount++;

                if ( currentNeighborCount == maxNeighborPerAtom )
                    return;
            }
        }
    }

    __global__ void findCirclesBetweenAtoms( const GridInfo         gridInfo,
                                             const uint32_t * const cellsStart,
                                             const uint32_t * const cellsEnd,
                                             SsesdfContext          sesContext )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= sesContext.atomNb )
            return;

        const float4 currentAtom = sesContext.getAtom( index );
        const int3   gridPos     = gridInfo.getGridPosition( make_float3( currentAtom ) );

        constexpr float MaxVdwRadius  = 3.48f;
        const float     gridCellRange = currentAtom.w + MaxVdwRadius + 2.f * sesContext.probeRadius;
        const int3      cellsInRange  = max( make_int3( ceilf( gridCellRange / gridInfo.cellSize ) ), make_int3( 1 ) );
        const int3      start         = max( gridPos - cellsInRange, make_int3( 0 ) );
        const int3      end           = min( gridPos + cellsInRange, gridInfo.size - 1 );

        uint32_t neighborCount = 0;
        for ( int z = start.z; z <= end.z; z++ )
        {
            for ( int y = start.y; y <= end.y; y++ )
            {
                for ( int x = start.x; x <= end.x; x++ )
                {
                    const int3     currentGridPosition = make_int3( x, y, z );
                    const uint32_t currentGridHash     = gridInfo.getHash( currentGridPosition );
                    const uint32_t startIndex          = cellsStart[ currentGridHash ];

                    if ( startIndex == AccelerationGrid::EmptyGridCellValue )
                        continue;

                    const uint32_t endIndex = cellsEnd[ currentGridHash ];

                    findCirclesInCell( index,
                                       currentAtom,
                                       sesContext.atoms,
                                       startIndex,
                                       endIndex,
                                       sesContext.probeRadius,
                                       sesContext.maxNeighborPerAtom,
                                       neighborCount,
                                       sesContext.neighborIds );
                    if ( neighborCount == sesContext.maxNeighborPerAtom )
                    {
                        printf( "Error: Too many neighbors found for atom %u.\n", blockIdx.x );

                        z = end.z + 1;
                        y = end.y + 1;
                        break;
                    }
                }
            }
        }

        sesContext.neighborNb[ index ] = neighborCount;
    }

    __global__ void computeCirclesVisibilityStatus( SsesdfContext sesContext )
    {
        extern __shared__ char sharedBuffer[];
        auto *                 blockAtomsData = reinterpret_cast<float4 *>( sharedBuffer );

        float4   blockAtom;
        uint16_t blockAtomCircleNb;
        uint32_t j;
        float4   jthData;
        if ( blockIdx.x < sesContext.atomNb )
        {
            blockAtom         = sesContext.getAtom( blockIdx.x );
            blockAtomCircleNb = sesContext.getNeighborNb( blockIdx.x );
            if ( threadIdx.x < blockAtomCircleNb )
            {
                j       = sesContext.getNeighborId( blockIdx.x, threadIdx.x );
                jthData = sesContext.getAtom( j );

                blockAtomsData[ threadIdx.x ] = jthData;
            }
        }

        cg::this_thread_block().sync();

        const uint32_t globalCircleIdx = blockIdx.x * sesContext.maxNeighborPerAtom + threadIdx.x;
        if ( threadIdx.x >= blockAtomCircleNb )
        {
            sesContext.visibilityStatus[ globalCircleIdx ] = 0;
            return;
        }

        uint32_t isVisible  = threadIdx.x < blockAtomCircleNb && blockIdx.x < j;
        uint32_t isComplete = threadIdx.x < blockAtomCircleNb && blockIdx.x < j;

        // Check if current circle is complete and not entirely occluded
        if ( isVisible )
        {
            const CircleGeometry circle = sesContext.getCircleGeometry( blockAtom, jthData );
            for ( uint16_t kCircleIndex = 0; kCircleIndex < blockAtomCircleNb; kCircleIndex++ )
            {
                if ( threadIdx.x == kCircleIndex )
                    continue;

                const float4 kData             = blockAtomsData[ kCircleIndex ];
                const float  kthExtendedRadius = kData.w + sesContext.probeRadius;

                const float3 kToCircle    = circle.center - make_float3( kData );
                const float  distToCircle = length( kToCircle );

                const float secondDistance = dot( kToCircle, circle.normal );
                const float cosTheta       = secondDistance / distToCircle;

                // Check if current circle is entirely covered by an atom
                const float sinTheta = ::sqrtf( 1.f - cosTheta * cosTheta );
                {
                    const float firstDistance    = sinTheta * distToCircle + circle.radius;
                    const float completeDistance = firstDistance * firstDistance + secondDistance * secondDistance;
                    if ( completeDistance - kthExtendedRadius * kthExtendedRadius < 1e-4f )
                    {
                        isVisible = 0;
                        break;
                    }
                }

                if ( isComplete )
                {
                    const float firstDistance    = ( -sinTheta ) * distToCircle + circle.radius;
                    const float completeDistance = firstDistance * firstDistance + secondDistance * secondDistance;
                    if ( completeDistance - kthExtendedRadius * kthExtendedRadius < 1e-4f )
                        isComplete = 0;
                }
            }

            sesContext.visibilityStatus[ globalCircleIdx ] = isVisible * ( 1 + isComplete );
        }
    }

    __global__ void fillCircles( SsesdfContext          sesContext,
                                 uint32_t               visibleCircleNb,
                                 const uint32_t * const trimmedToGlobalId,
                                 float4 *               positions,
                                 float4 *               axis,
                                 float4 *               visibilitySpheres )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= visibleCircleNb )
            return;

        const uint32_t globalId = trimmedToGlobalId[ index ];

        const Circle         ij     = sesContext.loadCircle( globalId );
        const float4         ai     = sesContext.getAtom( ij.i );
        const float4         aj     = sesContext.getAtom( ij.j );
        const CircleGeometry circle = sesContext.getCircleGeometry( ai, aj );

        // torus axis
        float3 ta = normalize( circle.center - make_float3( ai ) );
        // torus center
        float3 tc    = circle.center;
        float3 ortho = normalize( cross( ta, make_float3( 0.0f, 0.0f, 1.0f ) ) );

        // compute the tangential point X2 of the spheres
        float3 P = tc + ( ortho * circle.radius );
        float3 X = normalize( P - make_float3( ai ) ) * ai.w;
        float3 C = ( length( P - make_float3( ai ) )
                     / ( length( P - make_float3( aj ) ) + length( P - make_float3( ai ) ) ) )
                   * ( make_float3( aj ) - make_float3( ai ) );
        float distance = length( X - C );
        C              = ( C + make_float3( ai ) ) - tc;

        // write torus center & torus radius R
        positions[ index ] = make_float4( tc, circle.radius );
        // write torus axis & probe radius (= torus radius r)
        axis[ index ] = make_float4( ta, sesContext.probeRadius );
        // write visibility sphere
        visibilitySpheres[ index ] = make_float4( C, distance );
    }

    __global__ void fillIntersections( SsesdfContext sesContext,
                                       int4 *        intersectionAtomIds,
                                       float4 *      intersectionsPositions )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= *sesContext.dIntersectionNb )
            return;

        const int4 circlesIds = intersectionAtomIds[ index ];

        const uint32_t i = circlesIds.x / sesContext.maxNeighborPerAtom;
        const uint32_t j = sesContext.neighborIds[ circlesIds.x ];
        const uint32_t k = sesContext.neighborIds[ circlesIds.y ];

        atomicCAS( sesContext.visibilityStatus + circlesIds.x, 1, 2 );
        atomicCAS( sesContext.visibilityStatus + circlesIds.y, 1, 2 );
        atomicCAS( sesContext.visibilityStatus + circlesIds.z, 1, 2 );

        const float4 ai    = sesContext.getAtom( i );
        const float3 aiPos = make_float3( ai );
        float        sqR2  = ai.w + sesContext.probeRadius;
        sqR2 *= sqR2;

        const float4         aj = sesContext.getAtom( j );
        const CircleGeometry ij = sesContext.getCircleGeometry( ai, aj );

        const float4         ak = sesContext.getAtom( k );
        const CircleGeometry ik = sesContext.getCircleGeometry( ai, ak );

        const float3 u = ik.normal - dot( ik.normal, ij.normal ) * ij.normal;
        const float  t = dot( ik.center - ij.center, ik.normal ) / dot( u, ik.normal );

        const float3 x1              = ij.center + t * u;
        const float  squareIToX1Dist = length2( x1 - aiPos );

        const float c = fmax( -squareIToX1Dist + sqR2, 0.f );

        const float3 n            = normalize( cross( ij.normal, ik.normal ) );
        const float3 intersection = x1 + ( sign( static_cast<float>( circlesIds.w ) ) * sqrtf( c ) * n );

        intersectionsPositions[ index ] = make_float4( intersection, circlesIds.w );
        intersectionAtomIds[ index ]    = make_int4( i, j, k, 0 );
    }

    __global__ void findIntersectionsNeighbors( const GridInfo         gridInfo,
                                                SsesdfContext          sesContext,
                                                const float4 * const   sortedIntersectionPos,
                                                const uint32_t * const sortedIntersectionIds,
                                                const uint32_t * const cellsStart,
                                                const uint32_t * const cellsEnd,
                                                float4 *               intersectionPositions,
                                                int4 *                 intersectionAtomIds,
                                                float4 *               intersectionNeighbors )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;

        if ( index >= sesContext.intersectionWithNeighborNb )
            return;

        const uint32_t sortedIdx  = sortedIntersectionIds[ index ];
        const float3   currentPos = make_float3( sortedIntersectionPos[ index ] );
        const int3     gridPos    = gridInfo.getGridPosition( currentPos );

        const float gridCellRange = 2.f * sesContext.probeRadius;
        const int3  cellsInRange
            = max( make_int3( ceilf( make_float3( gridCellRange ) / gridInfo.cellSize ) ), make_int3( 1 ) );

        const int3 start = max( gridPos - cellsInRange, make_int3( 0 ) );
        const int3 end   = min( gridPos + cellsInRange, gridInfo.size - 1 );

        uint32_t neighborCount = 0;
        for ( int z = start.z; z <= end.z; z++ )
        {
            for ( int y = start.y; y <= end.y; y++ )
            {
                for ( int x = start.x; x <= end.x; x++ )
                {
                    const int3     currentGridPosition = make_int3( x, y, z );
                    const uint32_t currentGridHash     = gridInfo.getHash( currentGridPosition );
                    const uint32_t startIndex          = cellsStart[ currentGridHash ];

                    if ( startIndex == AccelerationGrid::EmptyGridCellValue )
                        continue;

                    const uint32_t endIndex = cellsEnd[ currentGridHash ];
                    for ( uint32_t j = startIndex; j < endIndex; j++ )
                    {
                        if ( index == j )
                            continue;

                        const float3 jthPos      = make_float3( sortedIntersectionPos[ j ] );
                        const float  currentDist = length( currentPos - jthPos );
                        if ( currentDist < gridCellRange )
                        {
                            intersectionNeighbors[ sortedIdx * sesContext.maxIntersectionNeighbors + neighborCount ]
                                = make_float4( jthPos, 0.f );

                            neighborCount++;
                            if ( neighborCount == sesContext.maxIntersectionNeighbors )
                            {
                                z = end.z + 1;
                                y = end.y + 1;
                                break;
                            }
                        }
                    }
                }
            }
        }

        intersectionPositions[ sortedIdx ].w = static_cast<int>( sortedIdx * sesContext.maxIntersectionNeighbors );
        intersectionAtomIds[ sortedIdx ].w   = static_cast<int>( neighborCount );
    }
} // namespace bcs::ssesdf
