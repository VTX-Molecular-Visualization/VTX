#ifdef _WIN32
#include <Windows.h>
#endif // _WIN32

#include <GL/gl3w.h>
#include <cuda_gl_interop.h>

#include "bcs/cuda/memory.cuh"

namespace bcs
{
    GraphicsDeviceBuffer::GraphicsDeviceBuffer( const std::size_t size, AccessType access, bool zeroInit ) :
        m_size( size ), m_registerFlags( hipGraphicsRegisterFlagsNone ), m_isInitialized( true )
    {
        assert( size > 0 && "Trying to allocate GraphicsDeviceBuffer of size 0" );
        glCreateBuffers( 1, &m_bufferId );

        GLsizeiptr accessMask = 0;
        if ( ( access & AccessType::Read ) != AccessType::None )
            accessMask |= GL_MAP_READ_BIT;
        if ( ( access & AccessType::Write ) != AccessType::None )
            accessMask |= GL_MAP_WRITE_BIT;

        glNamedBufferStorage( m_bufferId, size, nullptr, accessMask );
        cudaCheck( hipGraphicsGLRegisterBuffer( &m_binding, m_bufferId, m_registerFlags ) );

        if ( zeroInit )
        {
            const auto data = scopedMap<uint8_t>();
            cudaCheck( hipMemset( data.get(), 0, size ) );
        }
    }

    GraphicsDeviceBuffer::GraphicsDeviceBuffer( GraphicsDeviceBuffer && other ) noexcept :
        m_registerFlags( hipGraphicsRegisterFlagsNone )
    {
        std::swap( m_isInitialized, other.m_isInitialized );
        std::swap( m_size, other.m_size );
        std::swap( m_ptr, other.m_ptr );
        std::swap( m_accessType, other.m_accessType );
        std::swap( m_bufferId, other.m_bufferId );
        std::swap( m_type, other.m_type );
        std::swap( m_binding, other.m_binding );
        std::swap( m_registerFlags, other.m_registerFlags );
    }

    GraphicsDeviceBuffer & GraphicsDeviceBuffer::operator=( GraphicsDeviceBuffer && other ) noexcept
    {
        std::swap( m_isInitialized, other.m_isInitialized );
        std::swap( m_size, other.m_size );
        std::swap( m_ptr, other.m_ptr );
        std::swap( m_accessType, other.m_accessType );
        std::swap( m_bufferId, other.m_bufferId );
        std::swap( m_type, other.m_type );
        std::swap( m_binding, other.m_binding );
        std::swap( m_registerFlags, other.m_registerFlags );

        return *this;
    }

    GraphicsDeviceBuffer::~GraphicsDeviceBuffer()
    {
        if ( !m_isInitialized )
            return;

        unmap();
        cudaCheck( hipGraphicsUnregisterResource( m_binding ) );
        glDeleteBuffers( 1, &m_bufferId );
    }

    uint8_t * GraphicsDeviceBuffer::get( std::size_t byteOffset )
    {
        if ( !m_ptr )
        {
            cudaCheck( hipGraphicsMapResources( 1, &m_binding ) );
            cudaCheck(
                hipGraphicsResourceGetMappedPointer( reinterpret_cast<void **>( &m_ptr ), nullptr, m_binding ) );
        }

        return m_ptr + byteOffset;
    }

    uint8_t * GraphicsDeviceBuffer::get( std::size_t byteOffset ) const
    {
        if ( !m_ptr )
        {
            cudaCheck( hipGraphicsMapResources( 1, &m_binding ) );
            cudaCheck(
                hipGraphicsResourceGetMappedPointer( reinterpret_cast<void **>( &m_ptr ), nullptr, m_binding ) );
        }

        return m_ptr + byteOffset;
    }

    void GraphicsDeviceBuffer::unmap() const
    {
        if ( !m_ptr )
            return;

        m_ptr = nullptr;
        cudaCheck( hipGraphicsUnmapResources( 1, &m_binding ) );
    }
    GLuint GraphicsDeviceBuffer::getId() const { return m_bufferId; }

    std::size_t GraphicsDeviceBuffer::size() const { return m_size; }
    GraphicsDeviceBuffer::operator bool() const { return m_size > 0; }

    DeviceBuffer::DeviceBuffer( const std::size_t size, bool zeroInit ) : m_size( size )
    {
        cudaCheck( hipMallocAsync( reinterpret_cast<void **>( &m_ptr ), size, 0 ) );
        if ( zeroInit )
            cudaCheck( hipMemset( m_ptr, 0, size ) );

        m_initialized = true;
    }

    DeviceBuffer::DeviceBuffer( DeviceBuffer && other ) noexcept
    {
        std::swap( m_initialized, other.m_initialized );
        std::swap( m_size, other.m_size );
        std::swap( m_ptr, other.m_ptr );
    }

    DeviceBuffer & DeviceBuffer::operator=( DeviceBuffer && other ) noexcept
    {
        std::swap( m_initialized, other.m_initialized );
        std::swap( m_size, other.m_size );
        std::swap( m_ptr, other.m_ptr );

        return *this;
    }

    DeviceBuffer::~DeviceBuffer() { reset(); }

    void DeviceBuffer::reset()
    {
        if ( m_initialized )
        {
            cudaCheck( hipFreeAsync( m_ptr, 0 ) );
            m_ptr         = nullptr;
            m_initialized = false;
        }
    }

    DeviceBuffer::operator bool() const { return m_initialized; }

    std::size_t DeviceBuffer::size() const { return m_size; }

    ResultBuffer::ResultBuffer( const std::size_t size, bool zeroInit, bool graphics ) : m_isGraphics( graphics )
    {
        if ( graphics )
            m_graphicsBuffer = GraphicsDeviceBuffer( size, AccessType::ReadWrite, zeroInit );
        else
            m_buffer = DeviceBuffer( size, zeroInit );
    }

    ResultBuffer::ResultBuffer( ResultBuffer && other ) noexcept
    {
        std::swap( m_isGraphics, other.m_isGraphics );
        std::swap( m_graphicsBuffer, other.m_graphicsBuffer );
        std::swap( m_buffer, other.m_buffer );
    }

    ResultBuffer & ResultBuffer::operator=( ResultBuffer && other ) noexcept
    {
        std::swap( m_isGraphics, other.m_isGraphics );
        std::swap( m_graphicsBuffer, other.m_graphicsBuffer );
        std::swap( m_buffer, other.m_buffer );

        return *this;
    }

    ResultBuffer::~ResultBuffer() = default;

    void ResultBuffer::unmap() const
    {
        if ( m_isGraphics )
            return m_graphicsBuffer.unmap();
    }

    ResultBuffer::operator bool() const
    {
        if ( m_isGraphics )
            return m_graphicsBuffer;

        return m_buffer;
    }

    std::size_t ResultBuffer::size() const
    {
        if ( m_isGraphics )
            return m_graphicsBuffer.size();

        return m_buffer.size();
    }

    GLuint ResultBuffer::getId() const { return m_graphicsBuffer.getId(); }
} // namespace bcs
