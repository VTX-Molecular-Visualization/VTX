#include "hip/hip_runtime.h"
#include <>
#include <thrust/async/sort.h>
#include <thrust/sort.h>
#include <vector_functions.h>

#include "bcs/cuda/grid.cuh"
#include "bcs/cuda/math.cuh"
#include "bcs/cuda/setup.cuh"

namespace bcs
{
    // Based on NVIDIA particle simulation samples
    // Ref:
    // https://github.com/NVIDIA/cuda-samples/blob/master/Samples/2_Concepts_and_Techniques/particles/particles_kernel_impl.cuh
    __global__ void computeHashes( const GridInfo       gridInfo,
                                   const uint32_t       elementNb,
                                   const float4 * const positions,
                                   uint32_t *           hashes,
                                   uint32_t *           indices )
    {
        const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
        if ( index >= elementNb )
            return;

        const float4 p = positions[ index ];

        // get address in grid
        const uint32_t hash = gridInfo.getHash( make_float3( p ) );

        // store grid hash and particle index
        hashes[ index ]  = hash;
        indices[ index ] = index;
    }

    __global__ void buildAccelerationGrid( const uint32_t         elementNb,
                                           const uint32_t * const hashes,
                                           const uint32_t * const indices,
                                           const float4 * const   oldPos,
                                           uint32_t *             cellStarts,
                                           uint32_t *             cellEnds,
                                           float4 *               sortedPos )
    {
        extern __shared__ uint32_t sharedHash[]; // blockSize + 1 elements
        const uint32_t             index = blockIdx.x * blockDim.x + threadIdx.x;

        uint32_t hash;
        // handle case when no. of particles not multiple of block size
        if ( index < elementNb )
        {
            hash = hashes[ index ];

            // Load hash data into shared memory so that we can look
            // at neighboring particle's hash value without loading
            // two hash values per thread
            sharedHash[ threadIdx.x + 1 ] = hash;

            if ( index > 0 && threadIdx.x == 0 )
            {
                // first thread in block must load neighbor particle hash
                sharedHash[ 0 ] = hashes[ index - 1 ];
            }
        }

        __syncthreads();

        if ( index < elementNb )
        {
            // If this particle has a different cell index to the previous
            // particle then it must be the first particle in the cell,
            // so store the index of this particle in the cell.
            // As it isn't the first particle, it must also be the cell end of
            // the previous particle's cell

            const uint32_t lastHash = sharedHash[ threadIdx.x ];
            if ( index == 0 || hash != lastHash )
            {
                cellStarts[ hash ] = index;
                if ( index > 0 )
                    cellEnds[ lastHash ] = index;
            }

            if ( index == elementNb - 1 )
                cellEnds[ hash ] = index + 1;

            // Now use the sorted index to reorder the pos data
            const uint32_t sortedIndex = indices[ index ];
            sortedPos[ index ]         = oldPos[ sortedIndex ];
        }
    }

    AccelerationGrid::AccelerationGrid( GridInfo configuration, hipStream_t stream ) :
        m_configuration( configuration ),
        m_cellNb( configuration.size.x * configuration.size.y * configuration.size.z ),
        m_dCellsStart( DeviceBuffer::Typed<uint32_t>( m_cellNb, stream ) ),
        m_dCellsEnd( DeviceBuffer::Typed<uint32_t>( m_cellNb, stream ) )
    {
    }

    void AccelerationGrid::build( const uint32_t elementNb, const float4 * const positions )
    {
        hipMemset( m_dCellsStart.get<uint32_t>(), EmptyGridCellValue, sizeof( uint32_t ) * m_cellNb );

        // Re-allocatation only if needed
        if ( !m_dHashes || m_dHashes.size<uint32_t>() < elementNb )
            m_dHashes = DeviceBuffer::Typed<uint32_t>( elementNb );

        if ( !m_dIndices || m_dIndices.size<uint32_t>() < elementNb )
            m_dIndices = DeviceBuffer::Typed<uint32_t>( elementNb );

        if ( !m_dSortedPositions || m_dSortedPositions.size<float4>() < elementNb )
            m_dSortedPositions = DeviceBuffer::Typed<float4>( elementNb );

        auto [ numBlocks, numThreads ] = KernelConfig::From( elementNb, 256 );

        uint32_t * hashes  = m_dHashes.get<uint32_t>();
        uint32_t * indices = m_dIndices.get<uint32_t>();

        computeHashes<<<numBlocks, numThreads>>>( m_configuration, elementNb, positions, hashes, indices );
        cudaCheck( "Hashes computation failed" );

        thrust::sort_by_key( thrust::device, hashes, hashes + elementNb, indices );

        const uint32_t sharedMemorySize = sizeof( uint32_t ) * ( numThreads.x + 1 );
        buildAccelerationGrid<<<numBlocks, numThreads, sharedMemorySize>>>( elementNb,
                                                                            hashes,
                                                                            indices,
                                                                            positions,
                                                                            m_dCellsStart.get<uint32_t>(),
                                                                            m_dCellsEnd.get<uint32_t>(),
                                                                            m_dSortedPositions.get<float4>() );
        cudaCheck( "Acceleration grid construction failed" );
    }

} // namespace bcs
